#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cudnn.h>


// Block width for CUDA kernels
#define BW 128

#define tamanho_img 25
#define amostras 200

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != CUDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << cudnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)


/**
 * Computes ceil(x / y) for integral nonnegative values.
 */
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
    return (nominator + denominator - 1) / denominator;
}

/**
 * Computes the backpropagation results of the Softmax loss for each result in a batch.
 * Uses the softmax values obtained from forward propagation to compute the difference.
 *
 * @param label The training batch label values.
 * @param num_labels The number of possible labels.
 * @param batch_size The size of the trained batch.
 * @param diff The resulting gradient.
 */
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    const int label_value = static_cast<int>(label[idx]);

    // For each item in the batch, decrease the result of the label's value by 1
    diff[idx * num_labels + label_value] -= 1.0f;
}

struct FullyConnectedLayer
{
    int inputs, outputs;
    std::vector<float> pneurons, pbias;

    FullyConnectedLayer(int inputs_, int outputs_) : outputs(outputs_), inputs(inputs_),
        pneurons(inputs_ * outputs_), pbias(outputs_) {}
};

struct TrainingContext
{
    cudnnHandle_t cudnnHandle;
    hipblasHandle_t cublasHandle;

    cudnnTensorDescriptor_t dataTensor, l1Tensor, l2Tensor, l3Tensor;
    cudnnActivationDescriptor_t l1Activation, l2Activation, l3Activation;

    int m_gpuid;
    int m_batchSize;
    size_t m_workspaceSize;

    FullyConnectedLayer& ref_l1, &ref_l2, &ref_l3;

    // Disable copying
    TrainingContext& operator=(const TrainingContext&) = delete;
    TrainingContext(const TrainingContext&) = delete;

    TrainingContext(int gpuid, int batch_size,
                    FullyConnectedLayer& l1, FullyConnectedLayer& l2, FullyConnectedLayer& l3) : ref_l1(l1), ref_l2(l2), ref_l3(l3), m_gpuid(gpuid)
    {
        m_batchSize = batch_size;

        // Create CUBLAS and CUDNN handles
        checkCudaErrors(hipSetDevice(gpuid));
        checkCudaErrors(hipblasCreate(&cublasHandle));
        checkCUDNN(cudnnCreate(&cudnnHandle));

        // Create tensor descriptors
        checkCUDNN(cudnnCreateTensorDescriptor(&l1Tensor));
        checkCUDNN(cudnnCreateTensorDescriptor(&l2Tensor));
        checkCUDNN(cudnnCreateTensorDescriptor(&l3Tensor));

        checkCUDNN(cudnnCreateActivationDescriptor(&l1Activation));
        checkCUDNN(cudnnCreateActivationDescriptor(&l2Activation));
        //checkCUDNN(cudnnCreateActivationDescriptor(&l3Activation));

        
        // Set tensor descriptor sizes

        checkCUDNN(cudnnSetTensor4dDescriptor(l1Tensor,
                                              CUDNN_TENSOR_NCHW,
                                              CUDNN_DATA_FLOAT,
                                              batch_size, l1.outputs, 1, 1));

        checkCUDNN(cudnnSetTensor4dDescriptor(l2Tensor,
                                              CUDNN_TENSOR_NCHW,
                                              CUDNN_DATA_FLOAT,
                                              batch_size, l2.outputs, 1, 1));

        checkCUDNN(cudnnSetTensor4dDescriptor(l3Tensor,
                                              CUDNN_TENSOR_NCHW,
                                              CUDNN_DATA_FLOAT,
                                              batch_size, l3.outputs, 1, 1));

        checkCUDNN(cudnnSetActivationDescriptor(l1Activation, CUDNN_ACTIVATION_RELU,
                                                CUDNN_PROPAGATE_NAN, 0.0));

	checkCUDNN(cudnnSetActivationDescriptor(l2Activation, CUDNN_ACTIVATION_RELU,
                                                CUDNN_PROPAGATE_NAN, 0.0));
    }

    ~TrainingContext()
    {
        checkCudaErrors(hipSetDevice(m_gpuid));

        checkCudaErrors(hipblasDestroy(cublasHandle));
        checkCUDNN(cudnnDestroy(cudnnHandle));
        checkCUDNN(cudnnDestroyTensorDescriptor(dataTensor));
        checkCUDNN(cudnnDestroyTensorDescriptor(l1Tensor));
        checkCUDNN(cudnnDestroyTensorDescriptor(l2Tensor));
	checkCUDNN(cudnnDestroyTensorDescriptor(l3Tensor));
        checkCUDNN(cudnnDestroyActivationDescriptor(l1Activation));
        checkCUDNN(cudnnDestroyActivationDescriptor(l2Activation));
    }


    void ForwardPropagation(float *data, float *fc1, float *fc1relu,
                            float *fc2, float *fc2relu, float *fc3, float *result,
                            float *pfc1, float *pfc1bias,
                            float *pfc2, float *pfc2bias,
                            float *pfc3, float *pfc3bias, float *onevec)
    {        
        float alpha = 1.0f, beta = 0.0f;
        checkCudaErrors(hipSetDevice(m_gpuid));


        // FC1 layer
        // Forward propagate neurons using weights (fc1 = pfc1'*data)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    ref_l1.outputs, m_batchSize, ref_l1.inputs,
                                    &alpha,
                                    pfc1, ref_l1.inputs,
                                    data, ref_l1.inputs,
                                    &beta,
                                    fc1, ref_l1.outputs));
        // Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    ref_l1.outputs, m_batchSize, 1,
                                    &alpha,
                                    pfc1bias, ref_l1.outputs,
                                    onevec, 1,
                                    &alpha,
                                    fc1, ref_l1.outputs));

        // ReLU activation
        checkCUDNN(cudnnActivationForward(cudnnHandle, l1Activation, &alpha,
                                          l1Tensor, fc1, &beta, l1Tensor, fc1relu));

        // FC2 layer
        // Forward propagate neurons using weights (fc2 = pfc2'*fc1relu)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    ref_l2.outputs, m_batchSize, ref_l2.inputs,
                                    &alpha,
                                    pfc2, ref_l2.inputs,
                                    fc1relu, ref_l2.inputs,
                                    &beta,
                                    fc2, ref_l2.outputs));
        // Add bias using GEMM's "beta" (fc2 += pfc2bias*1_vec')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    ref_l2.outputs, m_batchSize, 1,
                                    &alpha,
                                    pfc2bias, ref_l2.outputs,
                                    onevec, 1,
                                    &alpha,
                                    fc2, ref_l2.outputs));

        // ReLU activation
        checkCUDNN(cudnnActivationForward(cudnnHandle, l2Activation, &alpha,
                                          l2Tensor, fc2, &beta, l2Tensor, fc2relu));

        // Forward propagate neurons using weights (fc3 = pfc3'*fc2relu)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                    ref_l3.outputs, m_batchSize, ref_l3.inputs,
                                    &alpha,
                                    pfc3, ref_l3.inputs,
                                    fc2relu, ref_l3.inputs,
                                    &beta,
                                    fc3, ref_l2.outputs));
        // Add bias using GEMM's "beta" (fc3 += pfc3bias*1_vec')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    ref_l3.outputs, m_batchSize, 1,
                                    &alpha,
                                    pfc3bias, ref_l3.outputs,
                                    onevec, 1,
                                    &alpha,
                                    fc3, ref_l3.outputs));



        // Softmax loss
        checkCUDNN(cudnnSoftmaxForward(cudnnHandle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL,
                                       &alpha, l3Tensor, fc3, &beta, l3Tensor, result));
    }

    void Backpropagation(float *data, float *labels, float *fc1, float *fc1relu,
                         float *fc2, float *fc2relu, float *fc3, float *fc3sfmx, float *dloss_data,
                         float *pfc1, float *pfc1bias,
                         float *pfc2, float *pfc2bias,
			 float *pfc3, float *pfc3bias,
                         float *gfc1, float *gfc1bias, float *dfc1, float *dfc1relu,
                         float *gfc2, float *gfc2bias, float *dfc2, float *dfc2relu,
			 float *gfc3, float *gfc3bias, float *dfc3,
                         void *workspace, float *onevec)
    {    
        float alpha = 1.0f, beta = 0.0f;

        float scalVal = 1.0f / static_cast<float>(m_batchSize);

        checkCudaErrors(hipSetDevice(m_gpuid));

        // Initialization (using the training error function)
        checkCudaErrors(hipMemcpyAsync(dloss_data, fc3sfmx, sizeof(float) * m_batchSize * ref_l3.outputs, hipMemcpyDeviceToDevice));
        
        // Softmax layer
        SoftmaxLossBackprop<<<RoundUp(m_batchSize, BW), BW>>>(labels, ref_l3.outputs, m_batchSize, dloss_data);

        // Accounting for batch size in SGD
        checkCudaErrors(hipblasSscal(cublasHandle, ref_l3.outputs * m_batchSize, &scalVal, dloss_data, 1));

        // FC3 layer
        // Compute derivative with respect to weights: gfc3 = (fc2relu * dfc3smax')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_l3.inputs, ref_l3.outputs, m_batchSize,
                                    &alpha, fc2relu, ref_l3.inputs, dloss_data, ref_l3.outputs, &beta, gfc3, ref_l3.inputs));
        // Compute derivative with respect to bias: gfc3bias = dfc3smax * 1_vec
        checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_l3.outputs, m_batchSize,
                                    &alpha, dloss_data, ref_l3.outputs, onevec, 1, &beta, gfc3bias, 1));
        // Compute derivative with respect to data (for previous layer): pfc3*dfc3smax (500x10*10xN)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_l3.inputs, m_batchSize, ref_l3.outputs,
                                    &alpha, pfc3, ref_l3.inputs, dloss_data, ref_l3.outputs, &beta, dfc3, ref_l3.inputs));
        
        // ReLU activation
        checkCUDNN(cudnnActivationBackward(cudnnHandle, l2Activation, &alpha,
                                           l2Tensor, fc2relu, l2Tensor, dfc3,
                                           l2Tensor, fc2, &beta, l2Tensor, dfc2relu));

        // FC2 layer
        // Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2relu')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_l2.inputs, ref_l2.outputs, m_batchSize,
                                    &alpha, fc1relu, ref_l2.inputs, dfc2relu, ref_l2.outputs, &beta, gfc2, ref_l2.inputs));
        // Compute derivative with respect to bias: gfc2bias = dfc2relu * 1_vec
        checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_l2.outputs, m_batchSize,
                                    &alpha, dfc2relu, ref_l2.outputs, onevec, 1, &beta, gfc2bias, 1));
        // Compute derivative with respect to data (for previous layer): pfc2*dfc2relu (800x500*500xN)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_l2.inputs, m_batchSize, ref_l2.outputs,
                                    &alpha, pfc2, ref_l2.inputs, dfc2relu, ref_l2.outputs, &beta, dfc2, ref_l2.inputs));

	        // ReLU activation
        checkCUDNN(cudnnActivationBackward(cudnnHandle, l1Activation, &alpha,
                                           l1Tensor, fc1relu, l1Tensor, dfc2,
                                           l1Tensor, fc1, &beta, l1Tensor, dfc1relu));


	 // FC1 layer
        // Compute derivative with respect to weights: gfc1 = (data * dfc1relu')
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_l1.inputs, ref_l1.outputs, m_batchSize,
                                    &alpha, data, ref_l1.inputs, dfc1relu, ref_l1.outputs, &beta, gfc1, ref_l1.inputs));
        // Compute derivative with respect to bias: gfc1bias = dfc1relu * 1_vec
        checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_l1.outputs, m_batchSize,
                                    &alpha, dfc1relu, ref_l1.outputs, onevec, 1, &beta, gfc1bias, 1));
        // Compute derivative with respect to data (for previous layer): pfc1*dfc1relu (800x500*500xN)
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_l1.inputs, m_batchSize, ref_l1.outputs,
                                    &alpha, pfc1, ref_l1.inputs, dfc1relu, ref_l1.outputs, &beta, dfc1, ref_l1.inputs));



    }

    void UpdateWeights(float learning_rate,
                       float *pfc1, float *pfc1bias,
                       float *pfc2, float *pfc2bias,
			float *pfc3, float *pfc3bias,
                       float *gfc1, float *gfc1bias,
                       float *gfc2, float *gfc2bias,
			float *gfc3, float *gfc3bias)
    {    
        float alpha = -learning_rate;

        checkCudaErrors(hipSetDevice(m_gpuid));

        // Fully connected 1
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l1.pneurons.size()),
                                    &alpha, gfc1, 1, pfc1, 1));
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l1.pbias.size()),
                                    &alpha, gfc1bias, 1, pfc1bias, 1));

        // Fully connected 2
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l2.pneurons.size()),
                                    &alpha, gfc2, 1, pfc2, 1));
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l2.pbias.size()),
                                    &alpha, gfc2bias, 1, pfc2bias, 1));

	// Fully connected 3
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l3.pneurons.size()),
                                    &alpha, gfc3, 1, pfc3, 1));
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_l3.pbias.size()),
                                    &alpha, gfc3bias, 1, pfc3bias, 1));
    }



};




int main() {
	int num_gpus;
	std::vector<float> dataset(tamanho_img*amostras);
	dataset = {1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,0,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
0,0,0,0,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,1,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,0,0,1,0,0,1,0,1,0,0,1,0,1,0,0,1,1,1,0,0,
1,0,0,1,1,1,0,1,0,0,1,1,0,0,0,1,0,1,0,0,1,0,0,1,1,
1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
1,0,0,0,1,1,1,0,1,1,1,1,0,1,1,1,0,1,0,1,1,0,1,0,1,
1,0,0,0,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,1,0,0,1,0,0,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,
1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,1,0,1,0,1,0,0,0,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,
1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,
1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
1,1,1,1,0,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
0,0,0,0,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,1,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,0,0,1,0,0,1,0,1,0,0,1,0,1,0,0,1,1,1,0,0,
1,0,0,1,1,1,0,1,0,0,1,1,0,0,0,1,0,1,0,0,1,0,0,1,1,
1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
1,0,0,0,1,1,1,0,1,1,1,1,0,1,1,1,0,1,0,1,1,0,1,0,1,
1,0,0,0,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,1,0,0,1,0,0,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,
1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,1,0,1,0,1,0,0,0,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,
1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,
1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
1,1,1,1,0,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
0,0,0,0,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,1,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,0,0,1,0,0,1,0,1,0,0,1,0,1,0,0,1,1,1,0,0,
1,0,0,1,1,1,0,1,0,0,1,1,0,0,0,1,0,1,0,0,1,0,0,1,1,
1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
1,0,0,0,1,1,1,0,1,1,1,1,0,1,1,1,0,1,0,1,1,0,1,0,1,
1,0,0,0,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,1,0,0,1,0,0,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,
1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,1,0,1,0,1,0,0,0,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
1,1,1,1,0,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
0,0,0,0,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,0,0,0,0,1,0,0,1,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,
1,1,1,1,1,0,0,1,0,0,1,0,1,0,0,1,0,1,0,0,1,1,1,0,0,
1,0,0,1,1,1,0,1,0,0,1,1,0,0,0,1,0,1,0,0,1,0,0,1,1,
1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,
1,0,0,0,1,1,1,0,1,1,1,1,0,1,1,1,0,1,0,1,1,0,1,0,1,
1,0,0,0,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,0,0,0,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,0,0,0,0,
1,1,1,1,1,1,1,0,0,1,1,0,1,0,1,1,0,0,1,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,0,1,0,0,1,0,0,1,1,
1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,
1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,0,1,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,1,0,1,0,1,0,0,0,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,
1,1,1,1,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,0,0,0,1,1,1,1,1,1,
1,0,0,0,1,0,1,0,1,0,0,0,1,0,0,0,0,1,0,0,0,0,1,0,0,
1,1,1,1,1,0,0,0,0,1,1,1,1,1,1,1,0,0,0,0,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,
1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,1,1,1,0,0,
0,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};

	std::vector<uint8_t> saidas(amostras);
	saidas = {1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,
	1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,
	1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,
	1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    	checkCudaErrors(hipGetDeviceCount(&num_gpus));
	printf("valor da contagem %d \n", num_gpus);
	FullyConnectedLayer l1(25, 5), l2(5,6), l3(6,5);

	std::random_device rd;
	
	float wfc1 = sqrt(3.0f / (l1.inputs * l1.outputs));
	printf("valor do aleatorio %.6f \n", wfc1);
        std::uniform_real_distribution<> dfc1(-wfc1, wfc1);

	float wfc2 = sqrt(3.0f / (l2.inputs * l2.outputs));
	printf("valor do aleatorio %.6f \n", wfc2);
        std::uniform_real_distribution<> dfc2(-wfc2, wfc2);

	float wfc3 = sqrt(3.0f / (l3.inputs * l3.outputs));
	printf("valor do aleatorio %.6f \n", wfc3);
        std::uniform_real_distribution<> dfc3(-wfc3, wfc3);
	
        for (auto&& iter : l1.pneurons)
            iter = static_cast<float>(0.5);
        for (auto&& iter : l1.pbias)
            iter = static_cast<float>(0.5);
        for (auto&& iter : l2.pneurons)
            iter = static_cast<float>(0.5);
        for (auto&& iter : l2.pbias)
            iter = static_cast<float>(0.5);
        for (auto&& iter : l3.pneurons)
            iter = static_cast<float>(0.5);
        for (auto&& iter : l3.pbias)
            iter = static_cast<float>(0.5);

float *d_data, *d_labels, *d_fc1, *d_fc1relu, *d_fc2, *d_fc2smax;


	
		

}

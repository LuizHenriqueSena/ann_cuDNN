#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>

#define tamanho_img 25
#define amostras 200

int main() {
	int num_gpus;
	std::vector<uint8_t> dataset(tamanho_img*amostras);
    	checkCudaErrors(hipGetDeviceCount(&num_gpus));
	printf("valor da contagem %d \n", num_gpus);

	

	
}
